
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>

#define BLOCKSIZE 16

__global__ void Cuda_Mult(int *d_a, int *d_b, int *d_res, int n){
    // dot product of two matrices 
    __shared__ int T1[BLOCKSIZE][BLOCKSIZE];
    __shared__ int T2[BLOCKSIZE][BLOCKSIZE];

    int R = blockIdx.y * BLOCKSIZE + threadIdx.y,C = blockIdx.x * BLOCKSIZE + threadIdx.x;
    int idx,Temp = 0;

    for (int i = 0; i < gridDim.x; ++i){
        idx = R * n + i * BLOCKSIZE + threadIdx.x;
        if(idx >= n*n){
            T1[threadIdx.y][threadIdx.x] = 0;
        }
        else{
            T1[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (i * BLOCKSIZE + threadIdx.y) * n + C;
        if(idx >= n*n){
            T2[threadIdx.y][threadIdx.x] = 0;
        }  
        else{
            T2[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCKSIZE; ++k) {
            Temp += T1[threadIdx.y][k] * T2[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(R < n && C < n){
        d_res[R * n + C] = Temp;
    }
}



void CPU_Mult(int *a, int *b, int *res, int n){
    // function to obtain dot product of two matrices
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int ans = 0.0;
            for (int h = 0; h < n; h++) {
                ans += a[i * n + h] * b[h * n + j];
            }
            res[i * n + j] = ans;
        }
    }
}


int main(int argc, char* argv[]){
    
    int option = 1;
    if(argc>1){   
        if(*argv[2] == 'S'){
            option = 0;
        }
    }
    //Fixed seed for illustration
    int N = 100;
    srand(1234);

    //allocate memory
    int *A, *B, *C, *new_C;
    hipHostMalloc((void **) &A, sizeof(int)*N*N, hipHostMallocDefault);
    hipHostMalloc((void **) &B, sizeof(int)*N*N, hipHostMallocDefault);
    hipHostMalloc((void **) &C, sizeof(int)*N*N, hipHostMallocDefault);
    
    // randomly initialize matrix A
    for (int i = 0; i < N; ++i){
		for (int j = 0; j < N; ++j){
			A[i*N + j] = (double)rand()/(double)(RAND_MAX/N);
		}
	}

    // randomly initialize matrix B
    for (int i = 0; i < N; ++i){
		for (int j = 0; j < N; ++j){
            B[i*N + j] = (double)rand()/(double)(RAND_MAX/N);
        }
    } 
    float gpu_elapsed_time;
    float cpu_elapsed_time;

    //event to calculate execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    if(option==1){
    	printf("To run serial mult on cpu use: %s -o S\n",argv[0]);

        //allocate space
		int *dev_A, *dev_B, *dev_C;
		hipMalloc((void **) &dev_A, sizeof(int)*N*N);
		hipMalloc((void **) &dev_B, sizeof(int)*N*N);
		hipMalloc((void **) &dev_C, sizeof(int)*N*N);

		hipEventRecord(start, 0);

        // copy matrix A and B from host to device mem
		hipMemcpy(dev_A, A, sizeof(int)*N*N, hipMemcpyHostToDevice);
		hipMemcpy(dev_B, B, sizeof(int)*N*N, hipMemcpyHostToDevice);

		unsigned int grid_Rs = (N + BLOCKSIZE - 1) / BLOCKSIZE;
		unsigned int gridev_Cs = (N + BLOCKSIZE - 1) / BLOCKSIZE;
		dim3 dimGrid(gridev_Cs, grid_Rs);
		dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);

        //Launch Kernal
		Cuda_Mult<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);    

        // Transefr results from device to host 
		hipMemcpy(C, dev_C, sizeof(int)*N*N, hipMemcpyDeviceToHost);

        //stop counting time
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

        //time for cuda evaluation
		hipEventElapsedTime(&gpu_elapsed_time, start, stop);
		printf("Time for mat mult of %dx%d . %dx%d on GPU: %f ms.\n\n", N, N, N, N, gpu_elapsed_time);

		hipFree(dev_A);
	    hipFree(dev_B);
	    hipFree(dev_C);
    }
    else{
    	hipHostMalloc((void **) &new_C, sizeof(int)*N*N, hipHostMallocDefault);

    	hipEventRecord(start, 0);
        //call normal multiplication by CPu. 
	    CPU_Mult(A, B, new_C, N);
	    
	    hipEventRecord(stop, 0);
	    hipEventSynchronize(stop);
	    
        //time for cpu evaluation
	    hipEventElapsedTime(&cpu_elapsed_time, start, stop);
	    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", N, N, N, N, cpu_elapsed_time);

	    hipHostFree(new_C);
    }
    
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    
    return 0;
}